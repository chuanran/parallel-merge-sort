#include "hip/hip_runtime.h"
float PerformParallelSort_WithoutSharedMemory(int n, int A[], int N, int I, int T, float* t)
{
	float totalTime_p;	
	float totalSortingTime;
	float totalCopyingTime;
	hipEvent_t* s_evt = new hipEvent_t[n];
	hipEvent_t* e_evt = new hipEvent_t[n];
	hipEvent_t startEvent_p, endEvent_p;
	hipEvent_t startCopy, endCopy;
	hipEvent_t startSort, endSort;
	hipError_t errorCode;
// Creating events for measuring performance
	hipEventCreate(&startEvent_p);
	hipEventCreate(&endEvent_p);
	hipEventCreate(&startCopy);
	hipEventCreate(&endCopy);
	hipEventCreate(&startSort);
	hipEventCreate(&endSort);
	for (int i = 0; i < n; i++)
	{
		hipEventCreate(&s_evt[i]);			
		hipEventCreate(&e_evt[i]);
	}
	errorCode = hipMemcpy(dev_a, A, sizeof(int)*N, hipMemcpyHostToDevice);
	ASSERT(errorCode);	

	hipEventRecord(startEvent_p);

	int* C = new int[N];
	
	// NumberOfElementsPerBlock = Shared memory size / 2
	// Sort<<<NumberOfBlocks, NumberOfThreadsPerBlock, SizeOfSharedMemory>>>
	//		(Source,Destination,NumberOfElementsPerBlock)

	DeviceProperties deviceProps;

	hipEventRecord(startSort);
	for (int i = 0; i < I; i++)
	{
		int n = 0;
		for (int d = 1; d <= (N/2); d *= 2)
		{
			hipEventRecord(s_evt[n]);
			int nThreads = min(T, N / (2*d));
			int numberOfBlocks = N / (2*d*nThreads);
			dim3 nBlocks = GetNBlocks(numberOfBlocks, deviceProps.MaxNumberOfBlocks);
			Sort_NonSharedMemory<<<nBlocks,nThreads>>>(dev_a, dev_c, d);
			hipDeviceSynchronize();
			errorCode = hipGetLastError();
			ASSERT(errorCode);
			
			// Swap contents of dev_c & dev_a
			SwapDeviceArrays();		
			hipEventRecord(e_evt[n]);
			hipEventSynchronize(e_evt[n]);
			hipEventElapsedTime(&t[n], s_evt[n], e_evt[n]);
			n++;
		}
	}
	hipEventRecord(endSort);
	hipEventSynchronize(endSort);
	hipEventElapsedTime(&totalSortingTime, startSort, endSort);
	
	hipEventRecord(startCopy);
	errorCode = hipMemcpy(C, dev_a, sizeof(int)*N, hipMemcpyDeviceToHost);
	hipEventRecord(endCopy);
	hipEventSynchronize(endCopy);
	hipEventElapsedTime(&totalCopyingTime, startCopy, endCopy);

	hipEventRecord(endEvent_p);
	hipEventSynchronize(endEvent_p);
	hipEventElapsedTime(&totalTime_p, startEvent_p, endEvent_p);

	//cout << totalCopyingTime << " [msec] for copying results to host." << endl;
	delete[] C;
	delete[] s_evt;
	delete[] e_evt;
	return totalTime_p / I;
}
